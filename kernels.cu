#include "hip/hip_runtime.h"
#include <iostream>
#include "kernels.cuh"

#define BLOCK_DIM 1024
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void twoSumKernel(int* data, int data_num, int target, int* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < data_num)
    {
        int current = data[idx];
        for (int i = idx + 1; i < data_num; i++)
        {
            if (current + data[i] == target)
            {
                out[0] = idx;
                out[1] = i;
            }
        }
    }
}

__global__ void twoSumKernel2(int* data, int data_num, int target, int* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr unsigned int warp_size = 32;
    constexpr unsigned int mask = 0xFFFFFFFF;
    if (idx < data_num)
    {
        int current = data[idx];
        for (int i = idx + 1; i < data_num + warp_size; i+=warp_size)
        {
            int current_data = i < data_num ? data[i] : 0;
            for (int j = 0; j<warp_size; j++)
            {
                int test = __shfl_sync(mask, current_data, j, warp_size);
                if (current + test == target)
                {
                    out[0] = idx;
                    out[1] = i;
                }
            }
        }
    }
}

__global__ void twoSumKernel3(int* data, int data_num, int target, int* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr unsigned int warp_size = 32;
    constexpr unsigned int mask = 0xFFFFFFFF;
    __shared__ int data_shared[BLOCK_DIM];

    if (idx < data_num)
    {
        int current = data[idx];
        for (int i = idx + 1; i < data_num + BLOCK_DIM; i+=BLOCK_DIM)
        {
            int current_data = i < data_num ? data[i] : 0;
            data_shared[threadIdx.x] = current_data;
            for (int j = 0; j<BLOCK_DIM; j+=warp_size)
            {
                // current_data=data_shared[(j+threadIdx.x)%BLOCK_DIM];
                current_data=data_shared[j + (threadIdx.x%warp_size)];
                for (int k = 0; k<warp_size; k++)
                {
                    int test = __shfl_sync(mask, current_data, k, warp_size);
                    if (current + test == target)
                    {
                        out[0] = idx;
                        out[1] = i;
                    }
                }
            }
        }
    }
}

extern "C" void twoSum(int* data, int* out, int target, int data_num, int variant)
{
    int* data_d;
    int* out_d;
    gpuErrchk(hipMalloc((void**) &data_d, data_num*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &out_d, 2*sizeof(int)));

    gpuErrchk(hipMemcpy(data_d, data, data_num*sizeof(int), hipMemcpyHostToDevice));

    dim3 block_size(BLOCK_DIM, 1, 1);
    dim3 grid_size(ceil((float)data_num/block_size.x), 1, 1);

    switch (variant)
    {
        case 1:
            twoSumKernel<<<grid_size, block_size>>>(data_d, data_num, target, out_d);
            break;
        case 2:
            twoSumKernel2<<<grid_size, block_size>>>(data_d, data_num, target, out_d);
            break;
        case 3:
            twoSumKernel3<<<grid_size, block_size>>>(data_d, data_num, target, out_d);
            break;
    }

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out, out_d, 2*sizeof(int), hipMemcpyDeviceToHost));
}
