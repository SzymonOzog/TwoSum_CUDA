#include "hip/hip_runtime.h"
#include <iostream>
#include "kernels.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void twoSumKernel(int* data, int data_num, int target, int* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < data_num)
    {
        int current = data[idx];
        for (int i = idx + 1; i < data_num; i++)
        {
            if (current + data[i] == target)
            {
                out[0] = idx;
                out[1] = i;
            }
        }
    }
}

extern "C" void twoSum(int* data, int* out, int target, int data_num)
{
    int* data_d;
    int* out_d;
    gpuErrchk(hipMalloc((void**) &data_d, data_num*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &out_d, 2*sizeof(int)));

    gpuErrchk(hipMemcpy(data_d, data, data_num*sizeof(int), hipMemcpyHostToDevice));

    dim3 block_size(1024, 1, 1);
    dim3 grid_size(ceil((float)data_num/block_size.x), 1, 1);

    twoSumKernel<<<grid_size, block_size>>>(data_d, data_num, target, out_d);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out, out_d, 2*sizeof(int), hipMemcpyDeviceToHost));
}
